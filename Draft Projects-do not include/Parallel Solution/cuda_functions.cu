#include "header.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// num of threads will be 3840 (num of cuda cores in our GPU) , each block will contain 32 threads, num of block will be 3840 / warp size (32)
int allocateDataOnDevice(Picture* device_Pics,Object* device_Objs,InputData* inputDataPtr)
{
    hipError_t err = hipSuccess;
    
	// Allocate and copy pictures
	int size = inputDataPtr->num_pictures * sizeof(Picture);
	err = hipMalloc((void **)&device_Pics, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory (error code %s)!\n",
	    hipGetErrorString(err));
		exit(EXIT_FAILURE);
		return 0;
	  }

	// copy data to device
	printf("Copying pictures from the host memory to the CUDA device\n");
	err = hipMemcpy(device_Pics, inputDataPtr->pictures, size, hipMemcpyHostToDevice);

	if (err != hipSuccess) {
	fprintf(stderr, "Failed to copy from host to device (error code %s)!\n",
    hipGetErrorString(err));
	exit(EXIT_FAILURE);
	return 0;
  }
  
  // Allocate and copy Objects
	size = inputDataPtr->num_objects * sizeof(Object);
	err = hipMalloc((void **)&device_Objs, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device memory (error code %s)!\n",
	    hipGetErrorString(err));
		exit(EXIT_FAILURE);
		return 0;
	  }

	// copy data to device
	printf("Copying objects from the host memory to the CUDA device\n");
	err = hipMemcpy(device_Pics, inputDataPtr->objects, size, hipMemcpyHostToDevice);

	if (err != hipSuccess) {
	fprintf(stderr, "Failed to copy from host to device (error code %s)!\n",
    hipGetErrorString(err));
	exit(EXIT_FAILURE);
	return 0;
  }
  
  return 1; // success
 	 
}
